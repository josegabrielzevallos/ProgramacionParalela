#include "hip/hip_runtime.h"

#include <sm_11_atomic_functions.h>
#include <iostream>
#include <math.h>
#include <time.h>

// File reading
#include <fstream>

// OpenCV headers
#include <cv.h>
#include <cxcore.h>
#include <highgui.h>

// CUDA headers
//#include <cutil_inline.h>

#define CONST_RADIUS 10
#define PI 3.141592653589793

#define BLOCK_SIZE 512
#define BLOCK_SIZE_HOUGH 360
#define STEP_SIZE 5
#define NUMBER_OF_STEPS 360/STEP_SIZE
#define GRAYLEVELS 256

// CUDA functions
extern "C" void Hough2D_CUDA(float* img, int width, int height, int radMin, int radMax, int* posX, int* posY, int* maxVal, int* resRad);

extern "C" void imadjustCUDA(unsigned char *inImg, unsigned char *outImg, int width, int height, float lowPerc, float highPerc);

extern "C" void adjustGammaCUDA(unsigned char *inImg, unsigned char *outImg, int width, int height, float gamma);



// Circ mask kernel storage
__constant__ int maskKernelX[NUMBER_OF_STEPS];
__constant__ int maskKernelY[NUMBER_OF_STEPS];

// Function to set precalculated relative coordinates for circle boundary coordinates
extern "C" void setMaskKernel(int *maskX, int *maskY)
{
	hipMemcpyToSymbol(HIP_SYMBOL(maskKernelX), maskX, NUMBER_OF_STEPS*sizeof(int));
	hipMemcpyToSymbol(HIP_SYMBOL(maskKernelY), maskY, NUMBER_OF_STEPS*sizeof(int));
}

// Kernel to set all pixel values to specified value
__global__ void setAllValuesKernel(int* houghSpace, int height, int width, float value)
{
	int const index = blockIdx.x * BLOCK_SIZE + threadIdx.x;
	if (index < height*width) {
		houghSpace[index] = value;
	}
	__syncthreads();
}

extern "C" void setAllValuesToCUDA(int* houghSpace, int height, int width, float value)
{
	//cout << "Setting all values to " << value << "..." << endl;
	dim3 dimGrid = (ceil((float)width*height/(float)BLOCK_SIZE));
	dim3 dimBlock = (BLOCK_SIZE);
	setAllValuesKernel<<<dimGrid, dimBlock>>>(houghSpace, height, width, value);
	hipDeviceSynchronize();
}

// Kernel to perform circular Hough transform
__global__ void houghTransformKernel(int* cudaHough, float* img, int height, int width, int radius)
{
	if (threadIdx.x < BLOCK_SIZE_HOUGH) {
		// Arrays to hold coordinates for circle pixels
		__shared__ float circVals[NUMBER_OF_STEPS];

		// There are 10 hough pixels calculated in each block
		int whichPixel = (int)threadIdx.x / NUMBER_OF_STEPS;
		// Calculate position for pixel in hough space
		int cpixIDy = (int)( ((float)(blockIdx.x*STEP_SIZE+whichPixel)) / (float)(width-(radius*2))) + radius;
		int cpixIDx = (blockIdx.x*10+whichPixel) % (width-(radius*2)) + radius;
		
		// Load image pixel from circle edge
		int xVal = cpixIDx + maskKernelX[threadIdx.x % NUMBER_OF_STEPS];
		int yVal = cpixIDy + maskKernelY[threadIdx.x % NUMBER_OF_STEPS];
		// Get the pixel value from the image
		float pixVal = img[yVal*width + xVal];
		//float pixVal = img[cpixIDy*width + cpixIDx]; // ## TO DELETE, INCORRECT

		//int houghVal = cudaHough[cpixIDy*width + cpixIDx];
		__syncthreads();

		if (pixVal > 0) {
			atomicAdd(cudaHough + cpixIDy*width + cpixIDx, 1);

		}
		__syncthreads();
	}

}

// Calls the Hough transform kernel
extern "C" void performHoughTransformCUDA(int* cudaHough, float* img, int height, int width, int radius)
{
	// Define grid and block dimensions
	dim3 dimGrid = ( ceil((float)(width-(2*radius)) * (height-(2*radius)) / (float)STEP_SIZE) );
	dim3 dimBlock = (BLOCK_SIZE_HOUGH);

	// Perform Hough transform and sync threads to get the final result
	houghTransformKernel<<<dimGrid, dimBlock>>>(cudaHough, img, height, width, radius);
	hipDeviceSynchronize();
} 

// Analyse the defined image area for circles using Hough Transform
extern "C" void Hough2D_CUDA(float* img, int width, int height, int radMin, int radMax, int* posX, int* posY, int* maxVal, int* resRad)
{
	int* houghSpace;
	houghSpace = (int*)malloc(width*height*sizeof(int));

	// Arrays for results
	int *posxArray, *posyArray, *maxValArray, *radArray;
	// Allocate correct memory for arrays
	posxArray = (int*)malloc((radMax-radMin)*sizeof(int));
	posyArray = (int*)malloc((radMax-radMin)*sizeof(int));
	maxValArray = (int*)malloc((radMax-radMin)*sizeof(int));
	radArray = (int*)malloc((radMax-radMin)*sizeof(int));

	// Allocate memory for CUDA images and matrices
	float *cudaImg;
	int *cudaHough;

	hipMalloc((void **)&cudaImg, width*height*sizeof(float));
	hipMalloc((void **)&cudaHough, width*height*sizeof(int));
	// Copy image from host to device
	hipMemcpy(cudaImg, img, width*height*sizeof(float), hipMemcpyHostToDevice);

	int ctrArr = 0, radius;
	for (int i=radMin; i < radMax; i++) {
		// Set all elements to zero
		setAllValuesToCUDA(cudaHough, height, width, 0);

		// Precalculate relX and relY
		radius = i;
		int ctr = 0;
		int* relX, *relY;
		relX = (int*)malloc(NUMBER_OF_STEPS*sizeof(int));
		relY = (int*)malloc(NUMBER_OF_STEPS*sizeof(int));
		for (int theta=0; theta < 360; theta+=STEP_SIZE) {
			// Calculate x and y coordinates
			float angle = (theta*PI) / 180;
			relX[ctr] = (int)(-radius*cos(angle));
			relY[ctr] = (int)(-radius*sin(angle));
			ctr++;
		}

		// Set mask coordinates for circle
		setMaskKernel(relX, relY);

		// Free memory
		free(relX);
		free(relY);

		//performHoughTransformCUDA(cudaHough, cudaImg, height, width, radius, relX, relY, angleNum);
		performHoughTransformCUDA(cudaHough, cudaImg, height, width, radius);

		// Copy matrix from device to host
		hipMemcpy(houghSpace, cudaHough, width*height*sizeof(float), hipMemcpyDeviceToHost);

		// Find max value in the houghSpace
		*maxVal = 0;
		int index;
		int tempPosX, tempPosY, tempMaxVal = 0;

		for (int y=0; y < height; y++) {
			for (int x=0; x < width; x++) {
				//index = radius*width*height + y*width + x;
				index = y*width + x;
				if (tempMaxVal < houghSpace[index]) {
					tempMaxVal = houghSpace[index];
					tempPosX = x;
					tempPosY = y;
				}
			}
		}

		// Write results to arrays
		posxArray[ctrArr] = tempPosX;
		posyArray[ctrArr] = tempPosY;
		maxValArray[ctrArr] = tempMaxVal;
		radArray[ctrArr] = i;

		cout << "Current (radius: " << i << ") MaxVal: " << maxValArray[ctrArr] << " ctr: " << ctrArr << endl;

		ctrArr++;
	} // end for

	// Find the maximum value from arrays
	*maxVal = 0;
	for (int j=0; j < (radMax-radMin); j++) {
		cout << "MaxValArray: " << maxValArray[j] << " " << posxArray[j] << " " << posyArray[j] << " " << radArray[j] << endl;
		if (*maxVal < maxValArray[j]) {
			*maxVal = maxValArray[j];
			*posX = posxArray[j];
			*posY = posyArray[j];
			*resRad = radArray[j];
		}
	}

	// Free cuda memory
	hipFree(cudaImg);
	hipFree(cudaHough);

	// Free array memory
	free(posxArray);
	free(posyArray);
	free(maxValArray);
	free(radArray);
}


// ##################
// #### IMADJUST ####
// ##################


__global__ void AdjustImageIntensityKernel(float *imgOut, float *imgIn, int width, int height, float lowin, float lowout, float scale)
{
    __shared__ float bufData[BLOCK_SIZE];

	// Get the index of pixel
	const int index = blockIdx.x * BLOCK_SIZE + threadIdx.x;

	// Load data to shared variable
	bufData[threadIdx.x] = imgIn[index];

	// Check that it's not out of bounds
	if (index < (height*width)) {
		
		// Find the according multiplier
		float tempLevel = ( bufData[threadIdx.x] - lowin)*scale + lowout;
		
		// Check that it's within required range
		if (tempLevel < 0) {
			bufData[threadIdx.x] = 0;
		}
		else if (tempLevel > 1) {
			bufData[threadIdx.x] = 1;
		}
		else {
			bufData[threadIdx.x] = tempLevel;
		}

		// Write data back
		imgOut[index] = bufData[threadIdx.x];
	}
	
	// Synchronise threads to have the whole image fully processed for output
	__syncthreads();
}

// Resize the image
__global__ void ImageScalingKernel(float *imgOut, float *imgIn, int width, int height)
{
	__shared__ float inData[BLOCK_SIZE];
	// Get the index of pixel
	const int index = blockIdx.x * BLOCK_SIZE + threadIdx.x;

	// Load data to shared variable
	inData[threadIdx.x] = imgIn[index];

	if ( index < (width*height) ) {
		imgOut[index] = inData[threadIdx.x] / (float)255; 
	}

	__syncthreads();
}

// the CUDA sample implementaiton can be used
void ImageHistogramCUDA(float *pSrc, int width, int height, int * imghist)
{
	const int GrayThres = 256;
	for (int i=0; i< GrayThres; i++) imghist[i] = 0; 
	for (int i=0; i< width*height; i++) {
		int level = (int) (pSrc[i]*255); 
		imghist[level]+=1; 
	}

}

// Strech limit
void ImageStretchLimitCUDA(float *pSrc, int width, int height,
					   float tol_low, float tol_high, float *low, float *high )
{
	const int GrayThres = 256;
	int imghist[256]; 

	double cdf[GrayThres], sum;
	int i;
	bool bLowFound=false, bHighFound=false;

	//histogram
	ImageHistogramCUDA(pSrc,width,height,imghist);	
	
	// the below segment can be implemented on CPU only; 
	//*************************************************
	// cdf
	cdf[0]=imghist[0];
	for (i=1;i<GrayThres;i++)
		cdf[i] = cdf[i-1] + imghist[i];		
	sum = cdf[GrayThres-1];
	for (i=0;i<GrayThres;i++)
		cdf[i] /= sum;

	// find low and high
	for (i=0;i<GrayThres;i++)
	{
		if (cdf[i]>=tol_low && (bLowFound == false))
		{
			*low = (float)(i);
			bLowFound = true;
		}

		if (cdf[i]>=tol_high && (bHighFound == false))
		{
			*high = (float)(i);
			bHighFound = true;
		}
	}
	// convert to range [0 1]
	*low /= (GrayThres-1);
	*high /= (GrayThres-1);

}

// Adjusts image intensity depending on the current gray levels of the image (histogram stretching)
extern "C" void imadjustCUDA(unsigned char *inImg, unsigned char *outImg, int width, int height, float lowPerc, float highPerc)
{
	const int grayLevels = 256;
    float lowin, highin;
	float *tempBuffer = new float[width*height];
	float *imgInput, *imgBuffer, *imgOutput;
	clock_t init, final_gpu;

	// Convert input image to float
	for (int i=0; i < (width*height); i++) {
		tempBuffer[i] = (float) inImg[i];
	}

	// ### ALLOCATE CUDA ARRAYS ###
	hipMalloc((void **)&imgInput, width * height * sizeof(float));
	hipMalloc((void **)&imgBuffer, width * height * sizeof(float));
	hipMalloc((void **)&imgOutput, width * height * sizeof(float));

	// ### COPY TO CUDA MEMORY ###
	hipMemcpy(imgInput, tempBuffer,  width * height * sizeof(float), hipMemcpyHostToDevice);

	// Get number of blocks
	int gridSize = ceil( (float)(height*width) / (float)BLOCK_SIZE );

	// Assign sizes
	dim3 blocks( gridSize );
	dim3 threads( BLOCK_SIZE );

	// Image scaling Kernel
	ImageScalingKernel<<<blocks, threads>>>(imgBuffer, imgInput, width, height);
	hipDeviceSynchronize();

	// Copy image buffer back to host memory (for ImageStretchLimit function)
	hipMemcpy(tempBuffer, imgBuffer, width * height * sizeof(float), hipMemcpyDeviceToHost);

	// find out the 1% pixel intensity value and set it to "low"
	// find out the 99% pixel intensiy valeu and set it to "high"
	//ImageStretchLimitCUDA(tempBuffer, width, height, 0.01f,0.99f,&lowin,&highin);
	ImageStretchLimitCUDA(tempBuffer, width, height, lowPerc,highPerc,&lowin,&highin);

	// Adjust image intensity
	float lowout = 0, highout = 1;
	float range = highin-lowin; 
	float rangeout = highout-lowout;  
	float scale = rangeout/range;

	printf("Adjusting image intensities on GPU (CUDA)...\n");
	// Start timer
	init = clock();
	// Call the adjust image intensity kernel
    AdjustImageIntensityKernel<<<blocks, threads>>>(imgOutput, imgBuffer, width, height, lowin, lowout, scale);
	hipDeviceSynchronize();

	// Copy the result back
	hipMemcpy(tempBuffer, imgOutput, width * height * sizeof(float), hipMemcpyDeviceToHost);

	// Take time
	final_gpu=clock()-init;
	printf("Time taken for imadjust on GPU (CUDA): %f sec\n", (double)final_gpu / ((double)CLOCKS_PER_SEC));

	// convert it back to unsigned char
	for (int i =0; i< width*height; i++) {
		outImg[i] = (unsigned char) (tempBuffer[i]*255);  
	}

	// Free memory
	hipFree(imgInput);
	hipFree(imgBuffer);
	hipFree(imgOutput);
}


// #####################
// #### ADJUSTGAMMA ####
// #####################


__global__ void AdjustGammaKernel(float *imgOut, float *imgIn, int width, int height, float gamma, float minVal, float maxVal)
{
    __shared__ float bufData[BLOCK_SIZE];

	// Get the index of pixel
	const int index = blockIdx.x * BLOCK_SIZE + threadIdx.x;

	// Load data to shared variable
	bufData[threadIdx.x] = imgIn[index];

	// Check that it's not out of bounds
	if (index < (height*width)) {
		
		// Find the according multiplier
		float tempLevel = ( bufData[threadIdx.x] - minVal) / maxVal;
		
		tempLevel = powf(tempLevel, (double)1/gamma);
		
		// Check that it's within required range
		if (tempLevel < 0) {
			bufData[threadIdx.x] = 0;
		}
		else if (tempLevel > 1) {
			bufData[threadIdx.x] = 1;
		}
		else {
			bufData[threadIdx.x] = tempLevel;
		}

		// Write data back
		imgOut[index] = bufData[threadIdx.x];
	}
	
	// Synchronise threads to have the whole image fully processed for output
	__syncthreads();
}

extern "C" void adjustGammaCUDA(unsigned char *inImg, unsigned char *outImg, int width, int height, float gamma)
{
	const int grayLevels = 256;
    float lowin, highin;
	float *tempBuffer = new float[width*height];
	float *imgInput, *imgOutput;
	clock_t init, final_gpu;
	
	float minVal = 1000, maxVal = 0;

	// Convert input image to float
	for (int i=0; i < (width*height); i++) {
		tempBuffer[i] = (float) inImg[i];
		
		// Calculate min and max values in the image ## CAN BE ADDED TO DO ON CUDA LATER ON ##
		if (minVal > tempBuffer[i]) {
			minVal = tempBuffer[i];
		}
		if (maxVal < tempBuffer[i]) {
			maxVal = tempBuffer[i];
		}
	}
	

	// ### ALLOCATE CUDA ARRAYS ###
	hipMalloc((void **)&imgInput, width * height * sizeof(float));
	hipMalloc((void **)&imgOutput, width * height * sizeof(float));

	// ### COPY TO CUDA MEMORY ###
	hipMemcpy(imgInput, tempBuffer,  width * height * sizeof(float), hipMemcpyHostToDevice);

	// Get number of blocks
	int gridSize = ceil( (float)(height*width) / (float)BLOCK_SIZE );

	// Assign sizes
	dim3 blocks( gridSize );
	dim3 threads( BLOCK_SIZE );

	printf("Adjusting gamma on GPU (CUDA)...\n");
	// Start timer
	init = clock();
	// Image scaling Kernel
	AdjustGammaKernel<<<blocks, threads>>>(imgOutput, imgInput, width, height, gamma, minVal, maxVal);
	hipDeviceSynchronize();

	// Copy the result back
	hipMemcpy(tempBuffer, imgOutput, width * height * sizeof(float), hipMemcpyDeviceToHost);

	// Take time
	final_gpu=clock()-init;
	printf("Time taken for gamma adjustment on GPU (CUDA): %f sec\n", (double)final_gpu / ((double)CLOCKS_PER_SEC));

	// convert it back to unsigned char
	for (int i =0; i< width*height; i++) {
		outImg[i] = (unsigned char) (tempBuffer[i]*255);  
	}

	// Free memory
	hipFree(imgInput);
	hipFree(imgOutput);
}
